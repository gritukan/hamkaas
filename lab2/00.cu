#include "hip/hip_runtime.h"
#include "common.h"

#include <iostream>
#include <vector>

__global__ void DoSomethingKernel(double* ptr, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index > n) {
        return;
    }

    double x = ptr[index];
    for (int i = 0; i < 1e4; i++) {
        x += 0.2;
        x = cos(x);
    }
    ptr[index] = x;
}

int main()
{
    constexpr int N = 10000;

    for (int i = 0; i < 10; i++) {
        std::vector<double> data(N, 0);
        for (auto& x : data) {
            for (int j = 0; j < 1000; j++) {
                x += 0.3;
                x = cos(x);
            }
        }

        double* gpuData;
        CUDA_CHECK_ERROR(hipMalloc(&gpuData, N * sizeof(double)));
        CUDA_CHECK_ERROR(hipMemcpy(gpuData, data.data(), N * sizeof(double), hipMemcpyHostToDevice));
    
        constexpr int ThreadsPerBlock = 1;
        DoSomethingKernel<<<(N + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock>>>(gpuData, N);
        CUDA_CHECK_KERNEL();

        CUDA_CHECK_ERROR(hipMemcpy(data.data(), gpuData, N * sizeof(double), hipMemcpyDeviceToHost));
        CUDA_CHECK_ERROR(hipFree(gpuData));
    }
}
