#include "hip/hip_runtime.h"
#include "common.h"

#include <iostream>
#include <vector>

__global__ void DoSomethingKernel(double* ptr)
{
    for (int i = 0; i < 1e5; i++) {
        *ptr += 0.2;
        *ptr = cos(*ptr);
    }
}

int main()
{
    constexpr int N = 1000;

    for (int i = 0; i < 10; i++) {
        std::vector<double> data(N, 0);
        for (auto& x : data) {
            for (int j = 0; j < 1000; j++) {
                x += 0.3;
                x = cos(x);
            }
        }

        double* gpuData;
        CUDA_CHECK_ERROR(hipMalloc(&gpuData, N * sizeof(double)));
        CUDA_CHECK_ERROR(hipMemcpy(gpuData, data.data(), N * sizeof(double), hipMemcpyHostToDevice));
    
        DoSomethingKernel<<<N, 1>>>(gpuData);
        CUDA_CHECK_KERNEL();

        CUDA_CHECK_ERROR(hipMemcpy(data.data(), gpuData, N * sizeof(double), hipMemcpyDeviceToHost));
        CUDA_CHECK_ERROR(hipFree(gpuData));
    }
}
