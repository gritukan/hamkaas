#include "hip/hip_runtime.h"
#include "common.h"

#include <cassert>
#include <vector>
#include <iostream>

__global__ void Kernel(int64_t* a, int i)
{
    a[i] = a[2 * i] + a[2 * i + 1];
}

void DoSlow(int64_t* a, int n)
{
    hipStream_t stream;
    CUDA_CHECK_ERROR(hipStreamCreate(&stream));

    CUDA_CHECK_ERROR(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));

    for (int i = n - 1; i >= 1; --i) {
        Kernel<<<1, 1, 0, stream>>>(a, i);
    }

    hipGraph_t graph;
    CUDA_CHECK_ERROR(hipStreamEndCapture(stream, &graph));

    hipGraphExec_t graphExec;
    CUDA_CHECK_ERROR(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

    for (int i = 0; i < 5; i++) {
        TCudaEventTimer timer;
        timer.Start();

        CUDA_CHECK_ERROR(hipGraphLaunch(graphExec, 0));
        CUDA_CHECK_ERROR(hipDeviceSynchronize());

        std::cout << "Slow: time=" << timer.Stop() << "ms" << std::endl;

        int64_t result;
        CUDA_CHECK_ERROR(hipMemcpy(&result, a + 1, sizeof(int64_t), hipMemcpyDeviceToHost));
        assert(result == int64_t(n) * (n + 1) / 2);
    }

    CUDA_CHECK_ERROR(hipGraphExecDestroy(graphExec));
    CUDA_CHECK_ERROR(hipGraphDestroy(graph));
    CUDA_CHECK_ERROR(hipStreamDestroy(stream));
}

void DoFast(int64_t* a, int n)
{
    hipGraph_t graph;
    CUDA_CHECK_ERROR(hipGraphCreate(&graph, 0));

    // Your code here: add nodes to the graph.

    hipGraphExec_t graphExec;
    CUDA_CHECK_ERROR(hipGraphInstantiate(&graphExec, graph, 0));

    for (int i = 0; i < 5; ++i) {
        TCudaEventTimer timer;
        timer.Start();

        CUDA_CHECK_ERROR(hipGraphLaunch(graphExec, 0));
        CUDA_CHECK_ERROR(hipDeviceSynchronize());

        std::cout << "Fast: time=" << timer.Stop() << "ms" << std::endl;

        int64_t result;
        CUDA_CHECK_ERROR(hipMemcpy(&result, a + 1, sizeof(int64_t), hipMemcpyDeviceToHost));
        assert(result == int64_t(n) * (n + 1) / 2);
    }

    CUDA_CHECK_ERROR(hipGraphExecDestroy(graphExec));
    CUDA_CHECK_ERROR(hipGraphDestroy(graph));
}

int main()
{
    constexpr int N = 1 << 17;
    std::vector<int64_t> a(N);
    for (int i = 0; i < N; ++i) {
        a[i] = i + 1;
    }

    int64_t* gpuA;
    CUDA_CHECK_ERROR(hipMalloc(&gpuA, 2 * N * sizeof(int64_t)));
    CUDA_CHECK_ERROR(hipMemcpy(gpuA + N, a.data(), N * sizeof(int64_t), hipMemcpyHostToDevice));

    DoSlow(gpuA, N);

    // Clear result.
    constexpr int64_t Zero = 0;
    CUDA_CHECK_ERROR(hipMemcpy(gpuA + 1, &Zero, sizeof(int64_t), hipMemcpyHostToDevice));

    // DoFast(gpuA, N);

    return 0;
}
