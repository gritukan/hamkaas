#include "hip/hip_runtime.h"
#include "common.h"

#include <cassert>
#include <vector>
#include <iostream>

__global__ void AddKernel(int64_t* a, int64_t* b, int64_t* c, int64_t n)
{
    int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) {
        return;
    }

    c[index] = a[index] + b[index];
}

void DoStream(int64_t* a, int64_t* b, int64_t* c, int64_t n)
{
    hipStream_t stream;
    CUDA_CHECK_ERROR(hipStreamCreate(&stream));

    int64_t* gpuA;
    CUDA_CHECK_ERROR(hipMalloc(&gpuA, n * sizeof(int64_t)));

    int64_t* gpuB;
    CUDA_CHECK_ERROR(hipMalloc(&gpuB, n * sizeof(int64_t)));

    int64_t* gpuC;
    CUDA_CHECK_ERROR(hipMalloc(&gpuC, n * sizeof(int64_t)));

    for (int i = 0; i < 5; i++) {
        TCudaEventTimer timer;
        timer.Start();

        CUDA_CHECK_ERROR(hipMemcpy(gpuA, a, n * sizeof(int64_t), hipMemcpyHostToDevice));
        CUDA_CHECK_ERROR(hipMemcpy(gpuB, b, n * sizeof(int64_t), hipMemcpyHostToDevice));

        constexpr int ThreadsPerBlock = 256;
        AddKernel<<<(n + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock, 0, stream>>>(gpuA, gpuB, gpuC, n);

        CUDA_CHECK_ERROR(hipMemcpy(c, gpuC, n * sizeof(int64_t), hipMemcpyDeviceToHost));

        std::cout << "Stream: time=" << timer.Stop() << "ms" << std::endl;
    }

    for (int64_t i = 0; i < n; ++i) {
        assert(c[i] == a[i] + b[i]);
    }

    CUDA_CHECK_ERROR(hipStreamSynchronize(stream));
    CUDA_CHECK_ERROR(hipStreamDestroy(stream));

    CUDA_CHECK_ERROR(hipFree(gpuA));
    CUDA_CHECK_ERROR(hipFree(gpuB));
    CUDA_CHECK_ERROR(hipFree(gpuC));
}

void DoGraph(int64_t* a, int64_t* b, int64_t* c, int64_t n)
{
    hipStream_t stream;
    CUDA_CHECK_ERROR(hipStreamCreate(&stream));

    int64_t* gpuA;
    CUDA_CHECK_ERROR(hipMalloc(&gpuA, n * sizeof(int64_t)));

    int64_t* gpuB;
    CUDA_CHECK_ERROR(hipMalloc(&gpuB, n * sizeof(int64_t)));

    int64_t* gpuC;
    CUDA_CHECK_ERROR(hipMalloc(&gpuC, n * sizeof(int64_t)));

    CUDA_CHECK_ERROR(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));

    constexpr int ThreadsPerBlock = 256;
    AddKernel<<<(n + ThreadsPerBlock - 1) / ThreadsPerBlock, ThreadsPerBlock, 0, stream>>>(gpuA, gpuB, gpuC, n);

    hipGraph_t graph;
    CUDA_CHECK_ERROR(hipStreamEndCapture(stream, &graph));

    hipGraphExec_t graphExec;
    CUDA_CHECK_ERROR(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

    for (int i = 0; i < 5; i++) {
        TCudaEventTimer timer;
        timer.Start();

        CUDA_CHECK_ERROR(hipMemcpy(gpuA, a, n * sizeof(int64_t), hipMemcpyHostToDevice));
        CUDA_CHECK_ERROR(hipMemcpy(gpuB, b, n * sizeof(int64_t), hipMemcpyHostToDevice));

        CUDA_CHECK_ERROR(hipGraphLaunch(graphExec, stream));

        CUDA_CHECK_ERROR(hipMemcpy(c, gpuC, n * sizeof(int64_t), hipMemcpyDeviceToHost));

        std::cout << "Graph: time=" << timer.Stop() << "ms" << std::endl;
    }

    CUDA_CHECK_ERROR(hipGraphExecDestroy(graphExec));
    CUDA_CHECK_ERROR(hipGraphDestroy(graph));
    CUDA_CHECK_ERROR(hipStreamDestroy(stream));

    CUDA_CHECK_ERROR(hipFree(gpuA));
    CUDA_CHECK_ERROR(hipFree(gpuB));
    CUDA_CHECK_ERROR(hipFree(gpuC));
}

int main()
{
    constexpr int N = 1 << 25;
    std::vector<int64_t> a(N);
    std::vector<int64_t> b(N);
    std::vector<int64_t> c(N);
    for (int i = 0; i < N; ++i) {
        a[i] = i + 1;
        b[i] = 3 * i - 17;
    }

    DoStream(a.data(), b.data(), c.data(), N);
    DoGraph(a.data(), b.data(), c.data(), N);

    return 0;
}
