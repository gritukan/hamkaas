#include "hip/hip_runtime.h"
#include "common.h"

#include <algorithm>
#include <iostream>
#include <vector>

template <int KernelSize, int BlockDimensionSize>
__global__ void MaxPoolingKernel(double* input, double* output, int n, int m)
{
    // Write your code here.
}

std::vector<std::vector<double>> MaxPoolingGpu(std::vector<std::vector<double>> input)
{
    int n = input.size();
    int m = input[0].size();

    double* gpuInput;
    double* gpuOutput;
    CUDA_CHECK_ERROR(hipMalloc(&gpuInput, n * m * sizeof(double)));
    CUDA_CHECK_ERROR(hipMalloc(&gpuOutput, n * m * sizeof(double)));

    for (int i = 0; i < n; i++) {
        CUDA_CHECK_ERROR(hipMemcpy(gpuInput + i * m, input[i].data(), m * sizeof(double), hipMemcpyHostToDevice));
    }

    constexpr int KernelSize = 4;
    constexpr int BlockDimensionSize = 16;

    dim3 blocks((n + BlockDimensionSize - 1) / BlockDimensionSize, (m + BlockDimensionSize - 1) / BlockDimensionSize);
    dim3 threads(BlockDimensionSize, BlockDimensionSize);
    MaxPoolingKernel<KernelSize, BlockDimensionSize><<<blocks, threads>>>(gpuInput, gpuOutput, n, m);
    CUDA_CHECK_KERNEL();

    std::vector<std::vector<double>> output(n, std::vector<double>(m));
    for (int i = 0; i < n; i++) {
        CUDA_CHECK_ERROR(hipMemcpy(output[i].data(), gpuOutput + i * m, m * sizeof(double), hipMemcpyDeviceToHost));
    }
    CUDA_CHECK_ERROR(hipFree(gpuInput));
    CUDA_CHECK_ERROR(hipFree(gpuOutput));

    return output;
}

std::vector<std::vector<double>> MaxPoolingCpu(std::vector<std::vector<double>> data)
{
    int n = data.size();
    int m = data[0].size();

    std::vector<std::vector<double>> result(n, std::vector<double>(m));
    for (int x = 0; x < n; ++x) {
        for (int y = 0; y < m; ++y) {
            double& max = result[x][y];
            for (int dx = 0; dx < 4; dx++) {
                for (int dy = 0; dy < 4; dy++) {
                    if (x + dx < n && y + dy < m) {
                        max = std::max(max, data[x + dx][y + dy]);
                    }
                }
            }
        }
    }

    return result;
}

bool DoTest(std::vector<std::vector<double>> data)
{
    auto gpuResult = MaxPoolingGpu(data);
    auto cpuResult = MaxPoolingCpu(data);
    if (gpuResult == cpuResult) {
        std::cerr << "Test passed (n = " << data.size() << ", m = " << data[0].size() << ")" << std::endl;
        return true;
    } else {
        std::cerr << "Test failed (n = " << data.size() << ", m = " << data[0].size() << ")" << std::endl;
        return false;
    }
}

int main()
{
    std::vector<std::vector<std::vector<double>>> testValues;
    auto addTest = [&] (int n, int m) {
        std::vector<std::vector<double>> data(n, std::vector<double>(m));
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < m; j++) {
                data[i][j] = 1.0 * rand() / RAND_MAX;
            }
        }

        testValues.push_back(data);
    };
    for (int x = 1; x <= 10; x++) {
        for (int y = 1; y <= 10; y++) {
            addTest(x, y);
        }
    }

    for (int x = 1; x <= 5; x++) {
        addTest(x, 1000);
        addTest(x, 999);
        addTest(x, 1001);
        addTest(1000, x);
        addTest(999, x);
        addTest(1001, x);
    }

    addTest(1000, 1000);
    addTest(999, 1000);
    addTest(1001, 1000);
    addTest(1000, 999);
    addTest(1000, 1000);
    addTest(1000, 1001);

    int passedTests = 0;
    for (auto data : testValues) {
        if (DoTest(data)) {
            passedTests++;
        }
    }

    if (passedTests == testValues.size()) {
        std::cerr << "All tests passed" << std::endl;
        return 0;
    } else {
        std::cerr << "Some tests failed" << std::endl;
        return 1;
    }
}
