#include "hip/hip_runtime.h"
#include <cstdint>

#include <cassert>
#include <cstdio>

#include "helpers.h"
#include "tensor.h"

namespace NHamKaas {

constexpr int64_t MaxThreadsPerBlock = 256;
constexpr int64_t MaxBlockCount = 65535;

#define FOR_ALL_FLOAT_TYPES(XX) \
    XX(float) \
    XX(double) \

template <class T>
__global__ void SumTensorsBroadcastKernel(
    const T* lhs,
    const T* rhs,
    T* output,
    int64_t* lhsShape,
    int64_t* rhsShape,
    int64_t dimensions,
    int64_t outputSize)
{
    int64_t indices[MaxDimensions];

    int64_t threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    for (int64_t lhsIndex = threadIndex; lhsIndex < outputSize; lhsIndex += gridDim.x * blockDim.x) {
        int64_t lhsIndexCopy = lhsIndex;

        for (int64_t i = dimensions - 1; i >= 0; --i) {
            indices[i] = lhsIndexCopy % lhsShape[i];
            lhsIndexCopy /= lhsShape[i];
        }

        int64_t rhsIndex = 0;
        for (int64_t i = 0; i < dimensions; ++i) {
            int64_t index = rhsShape[i] == 1 ? 0 : indices[i];
            rhsIndex = rhsIndex * rhsShape[i] + index;
        }

        output[lhsIndex] = lhs[lhsIndex] + rhs[rhsIndex];
    }
}

template <class T>
void SumTensorsBroadcast(
    hipStream_t stream,
    const T* lhs,
    const T* rhs,
    T* output,
    int64_t* lhsShape,
    int64_t* rhsShape,
    int64_t dimensions,
    int64_t outputSize)
{
    constexpr int64_t ThreadsPerBlock = 256;
    int64_t blocks = (outputSize + ThreadsPerBlock - 1) / ThreadsPerBlock;
    blocks = std::min(blocks, MaxBlockCount);

    SumTensorsBroadcastKernel<T><<<blocks, ThreadsPerBlock, 0, stream>>>(
        lhs,
        rhs,
        output,
        lhsShape,
        rhsShape,
        dimensions,
        outputSize);
}

#define INSTANTIATE(T) \
    template void SumTensorsBroadcast( \
        hipStream_t stream, \
        const T* lhs, \
        const T* rhs, \
        T* output, \
        int64_t* lhsShape, \
        int64_t* rhsShape, \
        int64_t dimensions, \
        int64_t outputSize);
FOR_ALL_FLOAT_TYPES(INSTANTIATE)
#undef INSTANTIATE

template <class T>
__global__ void ReLUKernel(
    const T* input,
    T* output,
    int64_t size)
{
    int64_t threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    for (int64_t index = threadIndex; index < size; index += gridDim.x * blockDim.x) {
        output[index] = input[index] > 0 ? input[index] : 0;
    }
}

template <class T>
void ReLU(
    hipStream_t stream,
    const T* input,
    T* output,
    int64_t size)
{
    constexpr int64_t ThreadsPerBlock = 256;
    int64_t blocks = (size + ThreadsPerBlock - 1) / ThreadsPerBlock;
    blocks = std::min(blocks, MaxBlockCount);

    ReLUKernel<T><<<blocks, ThreadsPerBlock, 0, stream>>>(input, output, size);
}

#define INSTANTIATE(T) \
    template void ReLU( \
        hipStream_t stream, \
        const T* input, \
        T* output, \
        int64_t size);
FOR_ALL_FLOAT_TYPES(INSTANTIATE)
#undef INSTANTIATE

template <class T>
__global__ void SiLUKernel(
    const T* input,
    T* output,
    int64_t size)
{
    int64_t threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    for (int64_t index = threadIndex; index < size; index += gridDim.x * blockDim.x) {
        output[index] = input[index] / (1 + exp(-input[index]));
    }
}

template <class T>
void SiLU(
    hipStream_t stream,
    const T* input,
    T* output,
    int64_t size)
{
    constexpr int64_t ThreadsPerBlock = 256;
    int64_t blocks = (size + ThreadsPerBlock - 1) / ThreadsPerBlock;
    blocks = std::min(blocks, MaxBlockCount);

    SiLUKernel<T><<<blocks, ThreadsPerBlock, 0, stream>>>(input, output, size);
}

#define INSTANTIATE(T) \
    template void SiLU( \
        hipStream_t stream, \
        const T* input, \
        T* output, \
        int64_t size);
FOR_ALL_FLOAT_TYPES(INSTANTIATE)
#undef INSTANTIATE

template <class T>
__global__ void RMSNormKernel(
    const T* input,
    const T* weights,
    T* output,
    int64_t size,
    T epsilon)
{
    assert(blockIdx.x == 0);

    __shared__ T blockSum[MaxThreadsPerBlock];
    __shared__ T sharedNorm;

    T localSum = 0;
    for (int64_t i = threadIdx.x; i < size; i += blockDim.x) {
        localSum += input[i] * input[i];
    }
    blockSum[threadIdx.x] = localSum;

    __syncthreads();

    if (threadIdx.x == 0) {
        T norm = 0;
        for (int64_t i = 0; i < blockDim.x; ++i) {
            norm += blockSum[i];
        }
        norm /= size;
        norm += epsilon;
        norm = 1.0 / sqrt(norm);
        sharedNorm = norm;
    }

    __syncthreads();

    T norm = sharedNorm;

    for (int64_t i = threadIdx.x; i < size; i += blockDim.x) {
        output[i] = weights[i] * (input[i] * norm);
    }
}

template <class T>
void RMSNorm(
    hipStream_t stream,
    const T* input,
    const T* weights,
    T* output,
    int64_t size,
    T epsilon)
{
    constexpr int64_t ThreadsPerBlock = 256;
    RMSNormKernel<T><<<1, ThreadsPerBlock, 0, stream>>>(input, weights, output, size, epsilon);
}

#define INSTANTIATE(T) \
    template void RMSNorm( \
        hipStream_t stream, \
        const T* input, \
        const T* weights, \
        T* output, \
        int64_t size, \
        T epsilon);
FOR_ALL_FLOAT_TYPES(INSTANTIATE)
#undef INSTANTIATE

template <class T>
__global__ void ComplexHadamardProductBroadcastKernel(
    const T* lhs,
    const T* rhs,
    T* output,
    int64_t* lhsShape,
    int64_t* rhsShape,
    int64_t dimensions,
    int64_t outputSize)
{
    int64_t indices[MaxDimensions];

    int64_t threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    for (int64_t lhsIndex = threadIndex; lhsIndex < outputSize / 2; lhsIndex += gridDim.x * blockDim.x) {
        int64_t lhsIndexCopy = lhsIndex;

        for (int64_t i = dimensions - 2; i >= 0; --i) {
            indices[i] = lhsIndexCopy % lhsShape[i];
            lhsIndexCopy /= lhsShape[i];
        }

        int64_t rhsIndex = 0;
        for (int64_t i = 0; i + 1 < dimensions; ++i) {
            int64_t index = rhsShape[i] == 1 ? 0 : indices[i];
            rhsIndex = rhsIndex * rhsShape[i] + index;
        }

        output[2 * lhsIndex] = lhs[2 * lhsIndex] * rhs[2 * rhsIndex] - lhs[2 * lhsIndex + 1] * rhs[2 * rhsIndex + 1];
        output[2 * lhsIndex + 1] = lhs[2 * lhsIndex] * rhs[2 * rhsIndex + 1] + lhs[2 * lhsIndex + 1] * rhs[2 * rhsIndex];
    }
}

template <class T>
void ComplexHadamardProductBroadcast(
    hipStream_t stream,
    const T* lhs,
    const T* rhs,
    T* output,
    int64_t* lhsShape,
    int64_t* rhsShape,
    int64_t dimensions,
    int64_t outputSize)
{
    constexpr int64_t ThreadsPerBlock = 256;
    int64_t blocks = (outputSize / 2 + ThreadsPerBlock - 1) / ThreadsPerBlock;
    blocks = std::min(blocks, MaxBlockCount);

    ComplexHadamardProductBroadcastKernel<T><<<blocks, ThreadsPerBlock, 0, stream>>>(
        lhs,
        rhs,
        output,
        lhsShape,
        rhsShape,
        dimensions,
        outputSize);
}

#define INSTANTIATE(T) \
    template void ComplexHadamardProductBroadcast( \
        hipStream_t stream, \
        const T* lhs, \
        const T* rhs, \
        T* output, \
        int64_t* lhsShape, \
        int64_t* rhsShape, \
        int64_t dimensions, \
        int64_t outputSize);
FOR_ALL_FLOAT_TYPES(INSTANTIATE)
#undef INSTANTIATE

template <class T>
__global__ void HadamardProductBroadcastKernel(
    const T* lhs,
    const T* rhs,
    T* output,
    int64_t* lhsShape,
    int64_t* rhsShape,
    int64_t dimensions,
    int64_t outputSize)
{
    int64_t indices[MaxDimensions];

    int64_t threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    for (int64_t lhsIndex = threadIndex; lhsIndex < outputSize; lhsIndex += gridDim.x * blockDim.x) {
        int64_t lhsIndexCopy = lhsIndex;

        for (int64_t i = dimensions - 1; i >= 0; --i) {
            indices[i] = lhsIndexCopy % lhsShape[i];
            lhsIndexCopy /= lhsShape[i];
        }

        int64_t rhsIndex = 0;
        for (int64_t i = 0; i < dimensions; ++i) {
            int64_t index = rhsShape[i] == 1 ? 0 : indices[i];
            rhsIndex = rhsIndex * rhsShape[i] + index;
        }

        output[lhsIndex] = lhs[lhsIndex] * rhs[rhsIndex];
    }
}

template <class T>
void HadamardProductBroadcast(
    hipStream_t stream,
    const T* lhs,
    const T* rhs,
    T* output,
    int64_t* lhsShape,
    int64_t* rhsShape,
    int64_t dimensions,
    int64_t outputSize)
{
    constexpr int64_t ThreadsPerBlock = 256;
    int64_t blocks = (outputSize + ThreadsPerBlock - 1) / ThreadsPerBlock;
    blocks = std::min(blocks, MaxBlockCount);

    HadamardProductBroadcastKernel<T><<<blocks, ThreadsPerBlock, 0, stream>>>(
        lhs,
        rhs,
        output,
        lhsShape,
        rhsShape,
        dimensions,
        outputSize);
}

#define INSTANTIATE(T) \
    template void HadamardProductBroadcast( \
        hipStream_t stream, \
        const T* lhs, \
        const T* rhs, \
        T* output, \
        int64_t* lhsShape, \
        int64_t* rhsShape, \
        int64_t dimensions, \
        int64_t outputSize);
FOR_ALL_FLOAT_TYPES(INSTANTIATE)
#undef INSTANTIATE

template <class T>
__global__ void SoftmaxKernel(
    const T* input,
    T* output,
    int64_t* prefixSizePtr,
    int64_t size,
    int64_t vectorSize)
{
    __shared__ T buffer[MaxThreadsPerBlock];

    int64_t prefixSize = *prefixSizePtr;

    for (int64_t vectorIndex = blockIdx.x; vectorIndex < size / vectorSize; vectorIndex += gridDim.x) {
        const T* in = input + vectorIndex * vectorSize;
        T* out = output + vectorIndex * vectorSize;

        if (threadIdx.x < prefixSize) {
            T max = in[threadIdx.x];
            for (int64_t index = threadIdx.x; index < prefixSize; index += blockDim.x) {
                max = max > in[index] ? max : in[index];
            }

            buffer[threadIdx.x] = max;
        }

        __syncthreads();

        if (threadIdx.x == 0) {
            T max = buffer[0];
            for (int64_t i = 1; i < prefixSize && i < blockDim.x; ++i) {
                max = max > buffer[i] ? max : buffer[i];
            }

            buffer[threadIdx.x] = max;
        }

        __syncthreads();

        T max = buffer[0];
        T sum = 0;
        for (int64_t index = threadIdx.x; index < prefixSize; index += blockDim.x) {
            sum += exp(in[index] - max);
        }

        buffer[threadIdx.x] = sum;

        __syncthreads();

        if (threadIdx.x == 0) {
            T sum = 0;
            for (int64_t i = 0; i < prefixSize && i < blockDim.x; ++i) {
                sum += buffer[i];
            }

            buffer[threadIdx.x] = sum;
        }

        __syncthreads();

        sum = buffer[0];

        for (int64_t index = threadIdx.x; index < vectorSize; index += blockDim.x) {
            if (index < prefixSize) {
                out[index] = exp(in[index] - max) / sum;
            } else {
                out[index] = in[index];
            }
        }

        __syncthreads();
    }
}

template <class T>
void SlicedSoftmax(
    hipStream_t stream,
    const T* input,
    T* output,
    int64_t* prefixSizePtr,
    int64_t size,
    int64_t vectorSize)
{
    constexpr int64_t ThreadsPerBlock = 256;

    int64_t blocks = std::min(MaxBlockCount, size / vectorSize);
    SoftmaxKernel<T><<<blocks, ThreadsPerBlock, 0, stream>>>(
        input,
        output,
        prefixSizePtr,
        size,
        vectorSize);
}

#define INSTANTIATE(T) \
    template void SlicedSoftmax( \
        hipStream_t stream, \
        const T* input, \
        T* output, \
        int64_t* prefixSizePtr, \
        int64_t size, \
        int64_t vectorSize);
FOR_ALL_FLOAT_TYPES(INSTANTIATE)
#undef INSTANTIATE

template <class T>
__global__ void ReplaceKernel(
    T* input,
    int64_t inputSize,
    const T* replacement,
    int64_t replacementSize,
    const int64_t* begin,
    const int64_t* end)
{
    int64_t threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    for (int64_t index = threadIndex; index < replacementSize; index += gridDim.x * blockDim.x) {
        input[index + *begin] = replacement[index];
    }
}

template <class T>
void ReplaceSlice(
    hipStream_t stream,
    T* input,
    int64_t inputSize,
    const T* replacement,
    int64_t replacementSize,
    const int64_t* begin,
    const int64_t* end)
{
    constexpr int64_t ThreadsPerBlock = 256;
    int64_t blocks = (replacementSize + ThreadsPerBlock - 1) / ThreadsPerBlock;
    blocks = std::min(blocks, MaxBlockCount);

    ReplaceKernel<T><<<blocks, ThreadsPerBlock, 0, stream>>>(
        input,
        inputSize,
        replacement,
        replacementSize,
        begin,
        end);
}

#define INSTANTIATE(T) \
    template void ReplaceSlice( \
        hipStream_t stream, \
        T* input, \
        int64_t inputSize, \
        const T* replacement, \
        int64_t replacementSize, \
        const int64_t* begin, \
        const int64_t* end);
FOR_ALL_FLOAT_TYPES(INSTANTIATE)
#undef INSTANTIATE

template <class T>
__global__ void PermuteKernel(
    const T* input,
    T* output,
    int64_t* inputShape,
    int64_t* outputShape,
    int64_t* permutation,
    int64_t dimensions,
    int64_t size)
{
    int64_t indices[MaxDimensions];

    int64_t threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    for (int64_t inputIndex = threadIndex; inputIndex < size; inputIndex += gridDim.x * blockDim.x) {
        int64_t inputIndexCopy = inputIndex;

        for (int64_t i = dimensions - 1; i >= 0; --i) {
            indices[i] = inputIndexCopy % inputShape[i];
            inputIndexCopy /= inputShape[i];
        }

        int64_t outputIndex = 0;
        for (int64_t i = 0; i < dimensions; ++i) {
            outputIndex = outputIndex * outputShape[i] + indices[permutation[i]];
        }

        output[outputIndex] = input[inputIndex];
    }
}

template <class T>
void Permute(
    hipStream_t stream,
    const T* input,
    T* output,
    int64_t* inputShape,
    int64_t* outputShape,
    int64_t* permutation,
    int64_t dimensions,
    int64_t size)
{
    constexpr int64_t ThreadsPerBlock = 256;
    int64_t blocks = (size + ThreadsPerBlock - 1) / ThreadsPerBlock;
    blocks = std::min(blocks, MaxBlockCount);

    PermuteKernel<T><<<blocks, ThreadsPerBlock, 0, stream>>>(
        input,
        output,
        inputShape,
        outputShape,
        permutation,
        dimensions,
        size);
}

#define INSTANTIATE(T) \
    template void Permute( \
        hipStream_t stream, \
        const T* input, \
        T* output, \
        int64_t* inputShape, \
        int64_t* outputShape, \
        int64_t* permutation, \
        int64_t dimensions, \
        int64_t size);
FOR_ALL_FLOAT_TYPES(INSTANTIATE)
#undef INSTANTIATE

} // namespace NHamKaas
